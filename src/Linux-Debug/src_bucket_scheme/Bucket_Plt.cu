#include "hip/hip_runtime.h"
//This file sets the grid for network interaction with plts. 
#include "SystemStructures.h"
#include "Bucket_Plt.h"
#include "System.h"

#include "functor_neighbor.h"
#include "functor_bucket_indexer.h"
#include "function_extend.h"

void init_plt_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	//always set bucket count. Update total if different. 
	domainParams.XBucketCount_plt_intc = ceil((domainParams.maxX - domainParams.minX) / domainParams.gridSpacing_plt_intc) + 1;
	domainParams.YBucketCount_plt_intc = ceil((domainParams.maxY - domainParams.minY) / domainParams.gridSpacing_plt_intc) + 1;
	domainParams.ZBucketCount_plt_intc = ceil((domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing_plt_intc) + 1;

	if ( (domainParams.XBucketCount_plt_intc * domainParams.YBucketCount_plt_intc * domainParams.ZBucketCount_plt_intc) != domainParams.totalBucketCount_plt_intc) {
		std::cout<<"resetting plt intct"<< std::endl;
        std::cout<<"x-bucket: "<< domainParams.XBucketCount_plt_intc<<std::endl;
		std::cout<<"y-bucket: "<< domainParams.YBucketCount_plt_intc<<std::endl;
		std::cout<<"z-bucket: "<< domainParams.ZBucketCount_plt_intc<<std::endl;
		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount_plt_intc = domainParams.XBucketCount_plt_intc * domainParams.YBucketCount_plt_intc * domainParams.ZBucketCount_plt_intc;
		std::cout<<"grid: "<< domainParams.gridSpacing_plt_intc << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount_plt_intc<<std::endl;

		auxVecs.keyBegin_plt_intc.resize(domainParams.totalBucketCount_plt_intc);
		auxVecs.keyEnd_plt_intc.resize(domainParams.totalBucketCount_plt_intc);
		
        //platelets
		auxVecs.keyPltBegin.resize(domainParams.totalBucketCount_plt_intc); 
		auxVecs.keyPltEnd.resize(domainParams.totalBucketCount_plt_intc); 
 
	}

	thrust::fill(auxVecs.keyBegin_plt_intc.begin(),auxVecs.keyBegin_plt_intc.end(),0);
	thrust::fill(auxVecs.keyEnd_plt_intc.begin(),auxVecs.keyEnd_plt_intc.end(),0);
	//platelets
	thrust::fill(auxVecs.keyPltBegin.begin(),auxVecs.keyPltBegin.end(),0);
	thrust::fill(auxVecs.keyPltEnd.begin(),auxVecs.keyPltEnd.end(),0);

};

//convert buckets into neighboring scheme
void extend_plt_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys_plt_intc) * 27;
	//platelets
	unsigned endIndexPltExpanded = (auxVecs.endIndexBucketPltKeys_plt_intc) * 27;

	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value_plt_intc.size();
	thrust::fill(auxVecs.id_bucket_expanded_plt_intc.begin(),auxVecs.id_bucket_expanded_plt_intc.end(),0);
	thrust::fill(auxVecs.id_value_expanded_plt_intc.begin(),auxVecs.id_value_expanded_plt_intc.end(),0);

	thrust::fill(auxVecs.idPlt_bucket_expanded.begin(),auxVecs.idPlt_bucket_expanded.end(),0);
	thrust::fill(auxVecs.idPlt_value_expanded.begin(),auxVecs.idPlt_value_expanded.end(),0);




	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/*
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys_plt_intc); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_plt_intc.begin(),
				auxVecs.id_value_plt_intc.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_plt_intc.begin(),
				auxVecs.id_value_expanded_plt_intc.begin())));

	thrust::counting_iterator<unsigned> countingBegin(0);
 
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_plt_intc.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_plt_intc.begin(),
				countingBegin)) + endIndexExpanded,
		
		auxVecs.id_bucket_expanded_plt_intc.begin(),
		functor_neighbor(
			domainParams.XBucketCount_plt_intc,
			domainParams.YBucketCount_plt_intc,
			domainParams.ZBucketCount_plt_intc)); 

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded_plt_intc.begin(),
		auxVecs.id_bucket_expanded_plt_intc.end(),
		auxVecs.id_value_expanded_plt_intc.begin());


	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded_plt_intc.begin(),
		auxVecs.id_bucket_expanded_plt_intc.end(), search_begin,
		search_begin + domainParams.totalBucketCount_plt_intc,
		auxVecs.keyBegin_plt_intc.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded_plt_intc.begin(),
		auxVecs.id_bucket_expanded_plt_intc.end(),search_begin,
		search_begin + domainParams.totalBucketCount_plt_intc,
		auxVecs.keyEnd_plt_intc.begin());

	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> pltfirst(27);
	/**
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/


	thrust::constant_iterator<unsigned> pltlast = pltfirst + (auxVecs.endIndexBucketPltKeys_plt_intc); // this is NOT numerical addition!

	expand(pltfirst, pltlast,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				auxVecs.idPlt_value_expanded.begin())));


	thrust::counting_iterator<unsigned> pltcountingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket_expanded.begin(),
				pltcountingBegin)) + endIndexPltExpanded,
			auxVecs.idPlt_bucket_expanded.begin(),
		functor_neighbor(
			domainParams.XBucketCount_plt_intc,
			domainParams.YBucketCount_plt_intc,
			domainParams.ZBucketCount_plt_intc));



	//unsigned pltnumberOfOutOfRange = thrust::count_if(auxVecs.idPlt_bucket_expanded.begin(),
	//	auxVecs.idPlt_bucket_expanded.end(), is_greater_than(domainParams.totalBucketCount) );
	//unsigned pltnumberInsideRange = endIndexPltExpanded - pltnumberOfOutOfRange;

	//unsigned endIndexPltSearch = endIndexPltExpanded - pltnumberOfOutOfRange;

	thrust::sort_by_key(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),
		auxVecs.idPlt_value_expanded.begin());
	
	thrust::counting_iterator<unsigned> pltsearch_begin(0);

	thrust::lower_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(), pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount_plt_intc,
		auxVecs.keyPltBegin.begin());

	thrust::upper_bound(auxVecs.idPlt_bucket_expanded.begin(),
		auxVecs.idPlt_bucket_expanded.end(),pltsearch_begin,
		pltsearch_begin + domainParams.totalBucketCount_plt_intc,
		auxVecs.keyPltEnd.begin());

	/*
	unsigned choice = 0;

	unsigned bucket = auxVecs.idPlt_bucket[choice];
	std::cout<<"bucketplt 0: "<< bucket<<std::endl;
	std::cout<<"plt pos: "<<pltInfoVecs.pltLocX[0]<<" "<<pltInfoVecs.pltLocY[0]<<" "<<pltInfoVecs.pltLocZ[0]<<std::endl;
	std::cout<<"key len: "<< auxVecs.keyBegin.size() << std::endl;
	unsigned begin = auxVecs.keyBegin[bucket];
	unsigned end = auxVecs.keyEnd[bucket];
	
	std::cout<<"from bucket scheme:"<<std::endl;
	for (unsigned i = begin; i < end; i++) {
		
		unsigned nbr = auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		}
	}*/

	/*
	std::cout<<"from all plt:"<<std::endl;
	for (unsigned i = 0; i < generalParams.maxNodeCount; i++) {
		unsigned nbr = i;//auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		} 
	}*/


}


//takes nodes and places in buckets.
void build_plt_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_plt_intc.begin(),
				auxVecs.id_value_plt_intc.begin())),
		functor_bucket_indexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount_plt_intc,
            domainParams.YBucketCount_plt_intc,
            domainParams.ZBucketCount_plt_intc,
			domainParams.gridSpacing_plt_intc));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value_plt_intc.begin(),
		auxVecs.id_value_plt_intc.begin() + generalParams.maxNodeCount,
		auxVecs.id_bucket_plt_intc.begin());

auxVecs.endIndexBucketKeys_plt_intc = generalParams.maxNodeCount;

	//platelets
	//std::cout<<"bucket platelet"<<std::endl;
	thrust::counting_iterator<unsigned> indexBucketBegin1(0);
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				pltInfoVecs.pltLocX.begin(),
				pltInfoVecs.pltLocY.begin(),
				pltInfoVecs.pltLocZ.begin(),
				indexBucketBegin1)) + generalParams.maxPltCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.idPlt_bucket.begin(),
				auxVecs.idPlt_value.begin())),
		functor_bucket_indexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount_plt_intc,
            domainParams.YBucketCount_plt_intc,
            domainParams.ZBucketCount_plt_intc,
			domainParams.gridSpacing_plt_intc));


	//std::cout<<"end bucket platelet"<<std::endl;
//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.idPlt_value.begin(),
		auxVecs.idPlt_value.end(),
		auxVecs.idPlt_bucket.begin());

auxVecs.endIndexBucketPltKeys_plt_intc = generalParams.maxPltCount;
 
};
