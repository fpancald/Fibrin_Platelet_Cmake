#include "hip/hip_runtime.h"
//This file sets the grid for network self interaction

#include "SystemStructures.h"
#include "Bucket_Net.h"
#include "System.h"

#include "functor_neighbor.h"
#include "functor_bucket_indexer.h"
#include "function_extend.h"



//take domain and discretize into square buckets of size gridspace
void init_dim_general(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocX.begin(), nodeInfoVecs.nodeLocX.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocY.begin(), nodeInfoVecs.nodeLocY.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end())));

	//platelets
	if (generalParams.maxPltCount != 0) {
		domainParams.pltminX = (*(thrust::min_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
		domainParams.pltmaxX = (*(thrust::max_element(pltInfoVecs.pltLocX.begin(), pltInfoVecs.pltLocX.end())));
		domainParams.pltminY = (*(thrust::min_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
		domainParams.pltmaxY = (*(thrust::max_element(pltInfoVecs.pltLocY.begin(), pltInfoVecs.pltLocY.end())));
		domainParams.pltminZ = (*(thrust::min_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
		domainParams.pltmaxZ = (*(thrust::max_element(pltInfoVecs.pltLocZ.begin(), pltInfoVecs.pltLocZ.end())));
	}
	else {
		domainParams.pltminX = minXTemp;
		domainParams.pltmaxX = maxXTemp;
		domainParams.pltminY = minYTemp;
		domainParams.pltmaxY = maxYTemp;
		domainParams.pltminZ = minZTemp;
		domainParams.pltmaxZ = maxZTemp;
	}

	double space = 0.0;
	domainParams.minX = min(minXTemp, domainParams.pltminX) - space;
	domainParams.maxX = max(maxXTemp, domainParams.pltmaxX) + space;
	domainParams.minY = min(minYTemp, domainParams.pltminY) - space;
	domainParams.maxY = max(maxYTemp, domainParams.pltmaxY) + space;
	domainParams.minZ = min(minZTemp, domainParams.pltminZ) - space;
	domainParams.maxZ = max(maxZTemp, domainParams.pltmaxZ) + space;
};

void init_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	unsigned padding = 1;
	if (generalParams.iterationCounter == 0) {
		padding = 2;
	}
	else {
		padding = 1;
	}
	
	//on the first iteration, we allocate more, we don't plan on using it. 
	//always set bucket count. Update total if different. 
	domainParams.XBucketCount_net_intc = padding * ceil((domainParams.maxX - domainParams.minX) / domainParams.gridSpacing_net_intc) + 1;
	domainParams.YBucketCount_net_intc = padding * ceil((domainParams.maxY - domainParams.minY) / domainParams.gridSpacing_net_intc) + 1;
	domainParams.ZBucketCount_net_intc = padding * ceil((domainParams.maxZ - domainParams.minZ) / domainParams.gridSpacing_net_intc) + 1;

	if ( (domainParams.XBucketCount_net_intc * domainParams.YBucketCount_net_intc * domainParams.ZBucketCount_net_intc) > domainParams.totalBucketCount_net_intc	) {
		std::cout<<"resetting grid for network interact" << std::endl;
		std::cout<<"x-bucket: "<< domainParams.XBucketCount_net_intc<<std::endl;
		std::cout<<"y-bucket: "<< domainParams.YBucketCount_net_intc<<std::endl;
		std::cout<<"z-bucket: "<< domainParams.ZBucketCount_net_intc<<std::endl;

		//double amount of buckets in case of resizing networks
		domainParams.totalBucketCount_net_intc = domainParams.XBucketCount_net_intc * domainParams.YBucketCount_net_intc * domainParams.ZBucketCount_net_intc;
		std::cout<<"grid: "<< domainParams.gridSpacing_net_intc << std::endl;
		std::cout<<"total bucket count: "<< domainParams.totalBucketCount_net_intc<<std::endl;

		std::cout<<"minX: " << domainParams.minX << std::endl;
		std::cout<<"maxX: " << domainParams.maxX << std::endl;
		std::cout<<"minY: " << domainParams.minY << std::endl;
		std::cout<<"maxY: " << domainParams.maxY << std::endl;
		std::cout<<"minZ: " << domainParams.minZ << std::endl;
		std::cout<<"maxZ: " << domainParams.maxZ << std::endl;

		auxVecs.keyBegin_net_intc.resize(domainParams.totalBucketCount_net_intc);
		auxVecs.keyEnd_net_intc.resize(domainParams.totalBucketCount_net_intc);
 
	}

	thrust::fill(auxVecs.keyBegin_net_intc.begin(),auxVecs.keyBegin_net_intc.end(),0);
	thrust::fill(auxVecs.keyEnd_net_intc.begin(),auxVecs.keyEnd_net_intc.end(),0);

};

//convert buckets into neighboring scheme
void extend_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.endIndexBucketKeys_net_intc) * 27;
	

	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value_net_intc.size();
	thrust::fill(auxVecs.id_bucket_expanded_net_intc.begin(),auxVecs.id_bucket_expanded_net_intc.end(),0);
	thrust::fill(auxVecs.id_value_expanded_net_intc.begin(),auxVecs.id_value_expanded_net_intc.end(),0);


	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/*
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.endIndexBucketKeys_net_intc); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_net_intc.begin(),
				auxVecs.id_value_net_intc.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				auxVecs.id_value_expanded_net_intc.begin())));

	thrust::counting_iterator<unsigned> countingBegin(0);
 
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				countingBegin)) + endIndexExpanded,
		
		auxVecs.id_bucket_expanded_net_intc.begin(),
		functor_neighbor(
			domainParams.XBucketCount_net_intc,
			domainParams.YBucketCount_net_intc,
			domainParams.ZBucketCount_net_intc)); 

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(),
		auxVecs.id_value_expanded_net_intc.begin());


	thrust::counting_iterator<unsigned> search_begin(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(), search_begin,
		search_begin + domainParams.totalBucketCount_net_intc,
		auxVecs.keyBegin_net_intc.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(),search_begin,
		search_begin + domainParams.totalBucketCount_net_intc,
		auxVecs.keyEnd_net_intc.begin());


	/*
	unsigned choice = 0;

	unsigned bucket = auxVecs.idPlt_bucket[choice];
	std::cout<<"bucketplt 0: "<< bucket<<std::endl;
	std::cout<<"plt pos: "<<pltInfoVecs.pltLocX[0]<<" "<<pltInfoVecs.pltLocY[0]<<" "<<pltInfoVecs.pltLocZ[0]<<std::endl;
	std::cout<<"key len: "<< auxVecs.keyBegin.size() << std::endl;
	unsigned begin = auxVecs.keyBegin[bucket];
	unsigned end = auxVecs.keyEnd[bucket];
	
	std::cout<<"from bucket scheme:"<<std::endl;
	for (unsigned i = begin; i < end; i++) {
		
		unsigned nbr = auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		}
	}*/

	/*
	std::cout<<"from all plt:"<<std::endl;
	for (unsigned i = 0; i < generalParams.maxNodeCount; i++) {
		unsigned nbr = i;//auxVecs.id_value_expanded[i];
		unsigned buck = auxVecs.id_bucket[nbr];
		double x_dist = pltInfoVecs.pltLocX[choice] - nodeInfoVecs.nodeLocX[nbr];
		double y_dist = pltInfoVecs.pltLocY[choice] - nodeInfoVecs.nodeLocY[nbr];
		double z_dist = pltInfoVecs.pltLocZ[choice] - nodeInfoVecs.nodeLocZ[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if (dist < 1.0){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		} 
	}*/


}


//takes nodes and places in buckets.
void build_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	PltInfoVecs& pltInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.nodeLocX.begin(),
				nodeInfoVecs.nodeLocY.begin(),
				nodeInfoVecs.nodeLocZ.begin(),
				indexBucketBegin)) + generalParams.maxNodeCount,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_net_intc.begin(),
				auxVecs.id_value_net_intc.begin())),
		functor_bucket_indexer(
			domainParams.minX, domainParams.maxX, domainParams.minY,
			domainParams.maxY, domainParams.minZ, domainParams.maxZ,
			domainParams.XBucketCount_net_intc,
			domainParams.YBucketCount_net_intc,
			domainParams.ZBucketCount_net_intc,
			domainParams.gridSpacing_net_intc));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value_net_intc.begin(),
		auxVecs.id_value_net_intc.begin() + generalParams.maxNodeCount,
		auxVecs.id_bucket_net_intc.begin());

auxVecs.endIndexBucketKeys_net_intc = generalParams.maxNodeCount;

 
};
