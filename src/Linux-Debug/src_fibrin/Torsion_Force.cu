#include "System.h"

#include "SystemStructures.h"

#include "Torsion_Force.h"

#include "functor_misc.h"

#include "functor_torsion.h"


void Torsion_Force(
	NodeInfoVecs& nodeInfoVecs,
	TorsionInfoVecs& torsionInfoVecs,
	GeneralParams& generalParams)  {
	
const double PI = 3.14159265358979323846;  
if (generalParams.totalTorsionCount>0) { 

		thrust::counting_iterator<unsigned> startTorsionIter(0);
		thrust::counting_iterator<unsigned> endTorsionIter(generalParams.totalTorsionCount);
 
		//for_each guarrantees order. This is needed for iter count and saving to torsion force vectors.
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					startTorsionIter,
					torsionInfoVecs.leftIndex.begin(),
					torsionInfoVecs.centerIndex.begin(),
					torsionInfoVecs.rightIndex.begin(),
					torsionInfoVecs.angleZero.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					endTorsionIter,
					torsionInfoVecs.leftIndex.end(),
					torsionInfoVecs.centerIndex.end(),
					torsionInfoVecs.rightIndex.end(),
					torsionInfoVecs.angleZero.end())),
			functor_torsion(
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
				thrust::raw_pointer_cast(torsionInfoVecs.forceX.data()),
				thrust::raw_pointer_cast(torsionInfoVecs.forceY.data()),
				thrust::raw_pointer_cast(torsionInfoVecs.forceZ.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.isNodeFixed.data()),
				generalParams.torsionStiffness,
				generalParams.maxNodeCount,
				generalParams.totalTorsionCount,
				PI));  

		//reduce by key to get forces.Notice leftIndex is 1/3rd the length of torsion.forceX
		//this vector will be sorted each iteration, so it needs to be recopied.
		thrust::copy(torsionInfoVecs.leftIndex.begin(), torsionInfoVecs.leftIndex.end(), torsionInfoVecs.tempTorIndices.begin());
		thrust::copy(torsionInfoVecs.centerIndex.begin(), torsionInfoVecs.centerIndex.end(), torsionInfoVecs.tempTorIndices.begin() + generalParams.totalTorsionCount);
		thrust::copy(torsionInfoVecs.rightIndex.begin(), torsionInfoVecs.rightIndex.end(), torsionInfoVecs.tempTorIndices.begin() + 2 * generalParams.totalTorsionCount);


		//key, then value. Each vector returns sorted		
		thrust::sort_by_key(torsionInfoVecs.tempTorIndices.begin(), torsionInfoVecs.tempTorIndices.begin() + 3 * generalParams.totalTorsionCount,
			thrust::make_zip_iterator(
				thrust::make_tuple(
					torsionInfoVecs.forceX.begin(),
					torsionInfoVecs.forceY.begin(),
					torsionInfoVecs.forceZ.begin())), thrust::less<unsigned>());


		thrust::fill(torsionInfoVecs.tempForceX.begin(), torsionInfoVecs.tempForceX.end(), 0);
		thrust::fill(torsionInfoVecs.tempForceY.begin(), torsionInfoVecs.tempForceY.end(), 0);
		thrust::fill(torsionInfoVecs.tempForceZ.begin(), torsionInfoVecs.tempForceZ.end(), 0);
		thrust::fill(torsionInfoVecs.reducedIds.begin(), torsionInfoVecs.reducedIds.end(), 0);

		unsigned endKey = thrust::get<0>(
			thrust::reduce_by_key(
				torsionInfoVecs.tempTorIndices.begin(), 
				torsionInfoVecs.tempTorIndices.begin() + 3*generalParams.totalTorsionCount,
			thrust::make_zip_iterator(
				thrust::make_tuple(
					torsionInfoVecs.forceX.begin(),
					torsionInfoVecs.forceY.begin(),
					torsionInfoVecs.forceZ.begin())),
			torsionInfoVecs.reducedIds.begin(),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					torsionInfoVecs.tempForceX.begin(),
					torsionInfoVecs.tempForceY.begin(),
					torsionInfoVecs.tempForceZ.begin())),
			thrust::equal_to<unsigned>(), CVec3Add())) - torsionInfoVecs.reducedIds.begin();//binary_pred, binary_op

		
		thrust::for_each(
			thrust::make_zip_iterator(//1st begin
				thrust::make_tuple(
					torsionInfoVecs.reducedIds.begin(),
					torsionInfoVecs.tempForceX.begin(),
					torsionInfoVecs.tempForceY.begin(),
					torsionInfoVecs.tempForceZ.begin())),
			thrust::make_zip_iterator(//1st end
				thrust::make_tuple(
					torsionInfoVecs.reducedIds.begin(),
					torsionInfoVecs.tempForceX.begin(),
					torsionInfoVecs.tempForceY.begin(),
					torsionInfoVecs.tempForceZ.begin())) + endKey,
			functor_add_UCVec3_CVec3(
				generalParams.maxNodeCount,
				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));

	}

	
}