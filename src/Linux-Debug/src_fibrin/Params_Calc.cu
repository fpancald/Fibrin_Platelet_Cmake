#include "SystemStructures.h"
#include "functor_misc.h"

#include "System.h"
#include "Params_Calc.h"
#include "functor_calc_strain_params.h"


void Params_Calc(
    WLCInfoVecs& wlcInfoVecs,
    NodeInfoVecs& nodeInfoVecs,
    GeneralParams& generalParams,
    PltInfoVecs& pltInfoVecs) {

		//count positive and negative strains for edges that are not added. If an edge is added, a zero is placed on that strain.
		//notice that each thread will count edges twice, so divide by two at the end
	
		thrust::fill(nodeInfoVecs.discretizedEdgeStrain.begin(), nodeInfoVecs.discretizedEdgeStrain.end(),0.0);
		thrust::fill(nodeInfoVecs.discretizedEdgeAlignment.begin(), nodeInfoVecs.discretizedEdgeAlignment.end(),0.0);	

		//copy current host information to device for strain calculation. 
		thrust::copy(nodeInfoVecs.hostEdgeLeft.begin(),
			nodeInfoVecs.hostEdgeLeft.begin() + generalParams.currentEdgeCount,
			nodeInfoVecs.deviceEdgeLeft.begin());

		thrust::copy(nodeInfoVecs.hostEdgeRight.begin(),
			nodeInfoVecs.hostEdgeRight.begin() + generalParams.currentEdgeCount,
			nodeInfoVecs.deviceEdgeRight.begin());

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.deviceEdgeLeft.begin(),
					nodeInfoVecs.deviceEdgeRight.begin())),
					 
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.deviceEdgeLeft.begin(),
					nodeInfoVecs.deviceEdgeRight.begin())) + generalParams.currentEdgeCount,
					
			//outputs discretized strain etc			
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.discretizedEdgeStrain.begin(),
					nodeInfoVecs.discretizedEdgeAlignment.begin())),
					
			functor_calc_strain_params(
				generalParams.originLinkCount,
				generalParams.originEdgeCount,
				generalParams.originNodeCount,
				generalParams.maxNodeCount,
				generalParams.maxNeighborCount,
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()) ));
		
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.sumForcesOnNode.begin(),//save vector
				
                functor_norm());

			//platelets
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						pltInfoVecs.pltForceX.begin(),
						pltInfoVecs.pltForceY.begin(),
						pltInfoVecs.pltForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						pltInfoVecs.pltForceX.begin(),
						pltInfoVecs.pltForceY.begin(),
						pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
				pltInfoVecs.sumForcesOnPlt.begin(),//save vector
				functor_norm());
};