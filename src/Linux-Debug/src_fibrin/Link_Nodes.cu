#include "hip/hip_runtime.h"

#include "Link_Nodes.h"

#include "SystemStructures.h"
#include "System.h"

#include "functor_de_link_nodes.h"
#include "functor_link_nodes.h"


void Link_Nodes(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

		//Default fill values at 0.
		thrust::fill(nodeInfoVecs.linksThreadMade.begin(),
			nodeInfoVecs.linksThreadMade.end(), 0);
		
	//	thrust::fill(nodeInfoVecs.delinksThreadMade.begin(),
	//		nodeInfoVecs.delinksThreadMade.end(), 0);

		thrust::fill(nodeInfoVecs.idMadeTempLeft.begin(),
				nodeInfoVecs.idMadeTempLeft.end(), 0);

		thrust::fill(nodeInfoVecs.idMadeTempRight.begin(),
				nodeInfoVecs.idMadeTempRight.end(), 0);


		//unsigned globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(),wlcInfoVecs.globalNeighbors.end(),is_less_than(generalParams.maxNodeCount));
		//std::cout<<"currentEdgeCount varpre: "<< generalParams.currentEdgeCount << std::endl;
		//std::cout<<"currentEdgeCount globalpre: "<< globalcount/2 << std::endl;

		thrust::counting_iterator<unsigned> counter(0);
		thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket_net_intc.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket_net_intc.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.linksThreadMade.begin(),//output
			functor_link_nodes(
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),

				thrust::raw_pointer_cast(auxVecs.id_value_expanded_net_intc.data()),
				thrust::raw_pointer_cast(auxVecs.keyBegin_net_intc.data()),
				thrust::raw_pointer_cast(auxVecs.keyEnd_net_intc.data()),

				generalParams.fiberDiameter,
				generalParams.maxNeighborCount,
				generalParams.maxNodeCount,

				generalParams.maxLinksPerIteration,
				thrust::raw_pointer_cast(nodeInfoVecs.idMadeTempLeft.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.idMadeTempRight.data()) ) );



		/*	for (unsigned i = 0; i < idMadeTempLeft.size(); i++) {
				unsigned varL = idMadeTempLeft[i];
				unsigned varR = idMadeTempRight[i];

				if ((varL != 0) || (varR != 0))
					std::cout<< varL << " " <<varR << std::endl;
			}
		unsigned begin = 479 * generalParams.maxNeighborCount;
		unsigned end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 479: "<< id <<std::endl;
			}
		}
		begin = 1004 * generalParams.maxNeighborCount;
		end = begin + generalParams.maxNeighborCount;
		for (unsigned i = begin; i < end; i++){
			unsigned id = wlcInfoVecs.globalNeighbors[i];
			if (id < generalParams.maxNodeCount){
				std::cout<<" 1004: "<< id <<std::endl;
			}
		}*/
	/*	thrust::counting_iterator<unsigned> counterDeLink(0);

		thrust::transform(
						counterDeLink,
						counterDeLink + generalParams.maxNodeCount,
				nodeInfoVecs.delinksThreadMade.begin(),
			functor_de_link_nodes(
				thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
				generalParams.maxNeighborCount,
				generalParams.maxNodeCount ) );
*/

		//add links made by above function. links are double counted since threads create links from a->b and b->a
		unsigned num_placed_links = ceil( thrust::reduce(nodeInfoVecs.linksThreadMade.begin(), 
			nodeInfoVecs.linksThreadMade.end(), 0, thrust::plus<unsigned>()) / 2 );

		//std::cout << " num_placed_links: " << num_placed_links << std::endl;


		//sort by increasing. Notice, the sorting must take place for the entire vector since threads write to different places
		thrust::sort_by_key( 
			nodeInfoVecs.idMadeTempLeft.begin(),nodeInfoVecs.idMadeTempLeft.end(),
			nodeInfoVecs.idMadeTempRight.begin(),thrust::greater<unsigned>() );

		thrust::stable_sort_by_key(
			nodeInfoVecs.idMadeTempRight.begin(),nodeInfoVecs.idMadeTempRight.end(),
			nodeInfoVecs.idMadeTempLeft.begin(), thrust::greater<unsigned>() );


		//the copy is not needed for the full vector. The only portion copied is 2 * num_placed_links since that many id's were set. 
		thrust::copy(nodeInfoVecs.idMadeTempRight.begin(), 
			nodeInfoVecs.idMadeTempRight.begin() + 2 * num_placed_links, 
			nodeInfoVecs.host_id_right.begin() );
		thrust::copy(nodeInfoVecs.idMadeTempLeft.begin(), 
			nodeInfoVecs.idMadeTempLeft.begin() + 2 * num_placed_links, 
			nodeInfoVecs.host_id_left.begin());



		//old code, keep here in case of issues. use for validation. 
		unsigned idL_init = nodeInfoVecs.host_id_left[0];
		unsigned idR_init = nodeInfoVecs.host_id_right[0];


		unsigned count = 0;
		//std::cout << "nodeInfoVecs.idMadeTempLeft.size(): " << nodeInfoVecs.idMadeTempLeft.size() << std::endl;
//		for (unsigned i = 1; i < nodeInfoVecs.idMadeTempLeft.size(); i++) {
//			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
//			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
//			unsigned idL = nodeInfoVecs.host_id_left[i];
//			unsigned idR = nodeInfoVecs.host_id_right[i];
//
//			if ((idL == idL_init) && (idR == idR_init)){
//				count +=1;
//			}
//			else {
//				count = 0;
//			}
//			//reset initial id's
//			idL_init = idL;
//			idR_init = idR;
//
//
//			if ( ((idL != 0) || (idR != 0) ) && (count == 1)) {
//
//				//count edges
//				std::cout<<"placing id: "<< idL<<" " << idR<<std::endl;
//
//				
//				nodeInfoVecs.hostEdgeLeft[generalParams.currentEdgeCount] = (idL);
//				nodeInfoVecs.hostEdgeRight[generalParams.currentEdgeCount] = (idR);
//				generalParams.currentEdgeCount += 1;
//			} 
//
//		} 
		//end old code
 
		idL_init = nodeInfoVecs.host_id_left[0];
		idR_init = nodeInfoVecs.host_id_right[0];
		count = 0;
		for (unsigned i = 1; i < 2 * num_placed_links; i++) {
			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
			unsigned idL = nodeInfoVecs.host_id_left[i];
			unsigned idR = nodeInfoVecs.host_id_right[i];

			if ((idL == idL_init) && (idR == idR_init)) {
				count += 1;
			}
			else {
				count = 0;
			}
			//reset initial id's
			idL_init = idL;
			idR_init = idR;


			if (((idL != 0) || (idR != 0)) && (count == 1)) {

				//count edges
				//std::cout << "placing id from tester: " << idL << " " << idR << std::endl;


				nodeInfoVecs.hostEdgeLeft[generalParams.currentEdgeCount] = (idL);
				nodeInfoVecs.hostEdgeRight[generalParams.currentEdgeCount] = (idR);
				generalParams.currentEdgeCount += 1;
			}

		}

	/*	unsigned globalcount = thrust::count_if(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), is_less_than(generalParams.maxNodeCount));

		unsigned linksmade = *(thrust::max_element(linksThreadMade.begin(), linksThreadMade.end() ));
		unsigned delinksmade = *(thrust::max_element(delinksThreadMade.begin(), delinksThreadMade.end() ));
		std::cout<<"max links made this iteration: "<< linksmade << std::endl;
		std::cout<<"max unlinks made this iteration: "<< delinksmade << std::endl;

		std::cout<<"currentEdgeCount var: "<< generalParams.currentEdgeCount << std::endl;
		std::cout<<"currentEdgeCount global "<< globalcount/2 << std::endl;

		unsigned temp= thrust::reduce(	wlcInfoVecs.currentNodeEdgeCountVector.begin(),
			wlcInfoVecs.currentNodeEdgeCountVector.end());
		std::cout<<"currentEdgeCount dev: "<< temp << std::endl;
	*/




};
