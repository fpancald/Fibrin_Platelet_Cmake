#include "SystemStructures.h"
#include "Plt_Arm_Plt_Force.h"
#include "System.h"
#include "functor_plt_arm_plt.h"


//tendril-like force
//The limit is plt_tndrl_intrct (small number)
//Force is applied to platelets
//We do not use this tndrl for imaging. 

void Plt_Arm_Plt_Force(
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {


	//for (unsigned i = 0; i < auxVecs.idPlt_bucket.size(); i++)
	//	std::cout<<"plt bucket pltonplt: "<<auxVecs.idPlt_bucket[i] << std::endl;
        
	thrust::counting_iterator<unsigned> counter(0);

    thrust::transform(
      	thrust::make_zip_iterator(
        	thrust::make_tuple(
				counter,
        		auxVecs.idPlt_bucket.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin(),
                pltInfoVecs.pltForceX.begin(),
                pltInfoVecs.pltForceY.begin(),
                pltInfoVecs.pltForceZ.begin())),
    thrust::make_zip_iterator(
        thrust::make_tuple(
				counter,
        		auxVecs.idPlt_bucket.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin(),
                pltInfoVecs.pltForceX.begin(),
                pltInfoVecs.pltForceY.begin(),
                pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
    thrust::make_zip_iterator(
      	thrust::make_tuple(
				//DOES NOT RESET FORCES
        		pltInfoVecs.pltForceX.begin(),
        		pltInfoVecs.pltForceY.begin(),
        		pltInfoVecs.pltForceZ.begin())),

        functor_plt_arm_plt(
            generalParams.plt_tndrl_intrct,
            generalParams.pltRForce,
            generalParams.pltForce,
            generalParams.pltR,

            generalParams.maxPltCount,
			generalParams.maxIdCountFlag,
			generalParams.pltrelease,

            thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
            thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
            thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data()),

            thrust::raw_pointer_cast(auxVecs.idPlt_value_expanded.data()),//plt neighbors
            thrust::raw_pointer_cast(auxVecs.keyPltBegin.data()),
            thrust::raw_pointer_cast(auxVecs.keyPltEnd.data()),
			
            thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeId.data()),
            thrust::raw_pointer_cast(pltInfoVecs.tndrlNodeType.data()) ));

	};