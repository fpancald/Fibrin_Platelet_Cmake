#include "Plt_Vol_Exc_Force.h"

#include <vector>
#include "SystemStructures.h"
#include "functor_plt_vol_exc.h"
#include "functor_misc.h"
#include "System.h"


//Call the plt force on nodes functor
//This functor applies force to nodes from platelets 
//as well as the self platelet from other platelets to create volume exclusion
//The interaction count is: plt_other_intrct. No imaging is done here. 
void Plt_Vol_Exc_Force(
  	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {

    	thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);

		
    	thrust::fill(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), generalParams.maxNodeCount);
		//thrust::fill(pltInfoVecs.nodeImagingConnection.begin(),pltInfoVecs.nodeImagingConnection.end(), generalParams.maxNodeCount);

		thrust::counting_iterator<unsigned> counter(0);

        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
   					auxVecs.idPlt_bucket.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())),
        	thrust::make_zip_iterator( 
        		thrust::make_tuple(
					counter,
    				auxVecs.idPlt_bucket.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin(),
					pltInfoVecs.pltForceX.begin(),
					pltInfoVecs.pltForceY.begin(),
					pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator( 
        	 thrust::make_tuple(
				 //DOES NOT RESET FORCE
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
             functor_plt_vol_exc(
                generalParams.plt_other_intrct,
                generalParams.pltRForce,
				generalParams.pltRAdhesion,
                generalParams.pltForce,
                generalParams.pltR,

                generalParams.maxPltCount,
                generalParams.fiberDiameter,
		        generalParams.maxNodeCount,
                generalParams.maxNeighborCount,

                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),

                thrust::raw_pointer_cast(auxVecs.id_value_expanded_plt_intc.data()),
                thrust::raw_pointer_cast(auxVecs.keyBegin_plt_intc.data()),
                thrust::raw_pointer_cast(auxVecs.keyEnd_plt_intc.data()),
				 
                thrust::raw_pointer_cast(auxVecs.idPlt_value_expanded.data()),
                thrust::raw_pointer_cast(auxVecs.keyPltBegin.data()),
                thrust::raw_pointer_cast(auxVecs.keyPltEnd.data()),

                thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
                thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data())) );

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());


		//reduce and apply force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.end(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op

		//apply force to network 
		//and check of node is in plt volume or not
        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        	functor_add_UCVec3_CVec3_pltVol(
				generalParams.maxNodeCount,
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.isNodeInPltVol.data())));
	}