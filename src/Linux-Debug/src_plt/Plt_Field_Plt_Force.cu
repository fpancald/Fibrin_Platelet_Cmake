#include "SystemStructures.h"
#include "Plt_Field_Plt_Force.h"
#include "functor_plt_field_plt.h"
#include "System.h"

//for a given platelet, apply force from other platelets
//Applies force to self. 
void Plt_Field_Plt_Force(
  	GeneralParams& generalParams,
  	PltInfoVecs& pltInfoVecs,
  	AuxVecs& auxVecs) {


	thrust::counting_iterator<unsigned> counter(0);

    thrust::transform(
      	thrust::make_zip_iterator( 
        	thrust::make_tuple(
				counter,
        		auxVecs.idPlt_bucket.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin(),
                pltInfoVecs.pltForceX.begin(),
                pltInfoVecs.pltForceY.begin(),
                pltInfoVecs.pltForceZ.begin())),
    thrust::make_zip_iterator(
        thrust::make_tuple(
				counter,
        		auxVecs.idPlt_bucket.begin(),
          		pltInfoVecs.pltLocX.begin(),
          		pltInfoVecs.pltLocY.begin(),
          		pltInfoVecs.pltLocZ.begin(),
                pltInfoVecs.pltForceX.begin(),
                pltInfoVecs.pltForceY.begin(),
                pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
    thrust::make_zip_iterator(
      	thrust::make_tuple(
				//DOES NOT RESET FORCES
        		pltInfoVecs.pltForceX.begin(),
        		pltInfoVecs.pltForceY.begin(),
        		pltInfoVecs.pltForceZ.begin())),

         functor_plt_field_plt(
             generalParams.plt_other_intrct,
             generalParams.pltRForce,
             generalParams.pltForce,
             generalParams.pltR,
             generalParams.maxPltCount,

             thrust::raw_pointer_cast(pltInfoVecs.pltLocX.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocY.data()),
             thrust::raw_pointer_cast(pltInfoVecs.pltLocZ.data()),

            // thrust::raw_pointer_cast(pltInfoVecs.pltForceX.data()),
            // thrust::raw_pointer_cast(pltInfoVecs.pltForceY.data()),
            // thrust::raw_pointer_cast(pltInfoVecs.pltForceZ.data()),

             thrust::raw_pointer_cast(auxVecs.idPlt_value_expanded.data()),//plt neighbors
             thrust::raw_pointer_cast(auxVecs.keyPltBegin.data()),
             thrust::raw_pointer_cast(auxVecs.keyPltEnd.data()) ) );
};