#include "SystemStructures.h"
#include "Plt_Field_Node_Force.h"
#include "functor_misc.h"
#include "functor_plt_field_node.h"
#include "System.h"

//for a given platelet, search for network nodes, pull and push them
//Force field-like mode

void Plt_Field_Node_Force(
  	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,
	GeneralParams& generalParams,
	PltInfoVecs& pltInfoVecs,
	AuxVecs& auxVecs) {


		thrust::fill(pltInfoVecs.nodeUnreducedForceX.begin(), pltInfoVecs.nodeUnreducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceY.begin(), pltInfoVecs.nodeUnreducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeUnreducedForceZ.begin(), pltInfoVecs.nodeUnreducedForceZ.end(), 0.0);

		thrust::fill(pltInfoVecs.nodeReducedForceX.begin(), pltInfoVecs.nodeReducedForceX.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceY.begin(), pltInfoVecs.nodeReducedForceY.end(), 0.0);
		thrust::fill(pltInfoVecs.nodeReducedForceZ.begin(), pltInfoVecs.nodeReducedForceZ.end(), 0.0);



        //Call the plt force on nodes functor
		thrust::counting_iterator<unsigned> counter(0);

        thrust::transform(
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
   					auxVecs.idPlt_bucket.begin(),
        			pltInfoVecs.pltLocX.begin(),
        			pltInfoVecs.pltLocY.begin(),
        			pltInfoVecs.pltLocZ.begin(),
        		 	pltInfoVecs.pltForceX.begin(),
        		 	pltInfoVecs.pltForceY.begin(),
        		 	pltInfoVecs.pltForceZ.begin())),
        	thrust::make_zip_iterator(
        		thrust::make_tuple(
					counter,
    				auxVecs.idPlt_bucket.begin(),
        		 	pltInfoVecs.pltLocX.begin(),
        		 	pltInfoVecs.pltLocY.begin(),
        		 	pltInfoVecs.pltLocZ.begin(), 
        		 	pltInfoVecs.pltForceX.begin(),
        		 	pltInfoVecs.pltForceY.begin(),
        		 	pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
         //save plt forces
         thrust::make_zip_iterator(
        	 thrust::make_tuple(
				 //DOES NOT RESET FORCE
        		 pltInfoVecs.pltForceX.begin(),
        		 pltInfoVecs.pltForceY.begin(),
        		 pltInfoVecs.pltForceZ.begin())),
             functor_plt_field_node(
                 generalParams.plt_other_intrct,
                 generalParams.pltRForce,
                 generalParams.pltForce,
                 generalParams.pltR,

                 generalParams.maxPltCount,
                 generalParams.fiberDiameter,
				 generalParams.maxNodeCount,

                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
                 thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceX.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceY.data()),
                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedForceZ.data()),

                 thrust::raw_pointer_cast(pltInfoVecs.nodeUnreducedId.data()),

                 thrust::raw_pointer_cast(auxVecs.id_value_expanded_plt_intc.data()),//network neighbors
                 thrust::raw_pointer_cast(auxVecs.keyBegin_plt_intc.data()),
                 thrust::raw_pointer_cast(auxVecs.keyEnd_plt_intc.data()) ) );

        //now call a sort by key followed by a reduce by key to figure out which nodes are have force applied.
        //then make a functor that takes the id and force (4 tuple) and takes that force and adds it to the id'th entry in nodeInfoVecs.nodeForceX,Y,Z
        thrust::sort_by_key(pltInfoVecs.nodeUnreducedId.begin(), pltInfoVecs.nodeUnreducedId.end(),
        			thrust::make_zip_iterator(
        				thrust::make_tuple(
        					pltInfoVecs.nodeUnreducedForceX.begin(),
        					pltInfoVecs.nodeUnreducedForceY.begin(),
        					pltInfoVecs.nodeUnreducedForceZ.begin())), thrust::less<unsigned>());

 //   thrust::copy(pltInfoVecs.nodeUnreducedId.begin(),pltInfoVecs.nodeUnreducedId.end(), pltInfoVecs.nodeImagingConnection.begin());

 //   pltInfoVecs.numConnections = thrust::count_if(
  //      pltInfoVecs.nodeImagingConnection.begin(),
  //      pltInfoVecs.nodeImagingConnection.end(), is_less_than(generalParams.maxNodeCount) );


		//reduce network force
 		unsigned endKey = thrust::get<0>(
 			thrust::reduce_by_key(
 				pltInfoVecs.nodeUnreducedId.begin(),
 				pltInfoVecs.nodeUnreducedId.end(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(
 					pltInfoVecs.nodeUnreducedForceX.begin(),
 					pltInfoVecs.nodeUnreducedForceY.begin(),
 					pltInfoVecs.nodeUnreducedForceZ.begin())),
 			pltInfoVecs.nodeReducedId.begin(),
 			thrust::make_zip_iterator(
 				thrust::make_tuple(//need t check
 					pltInfoVecs.nodeReducedForceX.begin(),
 					pltInfoVecs.nodeReducedForceY.begin(),
 					pltInfoVecs.nodeReducedForceZ.begin())),
 			thrust::equal_to<unsigned>(), CVec3Add())) - pltInfoVecs.nodeReducedId.begin();//binary_pred, binary_op

		//apply force to network
        thrust::for_each(
        	thrust::make_zip_iterator(//1st begin
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())),
        	thrust::make_zip_iterator(//1st end
        		thrust::make_tuple(
        			pltInfoVecs.nodeReducedId.begin(),
        			pltInfoVecs.nodeReducedForceX.begin(),
        			pltInfoVecs.nodeReducedForceY.begin(),
        			pltInfoVecs.nodeReducedForceZ.begin())) + endKey,
        	functor_add_UCVec3_CVec3(
				generalParams.maxNodeCount,
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
        		thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data())));
};